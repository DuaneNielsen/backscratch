#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <assert.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

typedef struct {
  int width;
  int height;
  float * elements;
} Matrix;


Matrix initMatrix(int height, int width) {
  Matrix A;
  A.width = width;
  A.height = height;
  A.elements = (float*)malloc(width * height * sizeof(float));
  return A;
}

void setRandom(Matrix A) {
  for (int i = 0; i < A.height; i++)
    for (int j = 0; j < A.width; j++)
      A.elements[i*A.width + j] = (float)(rand() % 3);
}

void printMatrix(Matrix A){
  for (int i = 0; i < A.height; i++)
    for(int j = 0; j < A.width; j++) {
	  if ( j == 0 ) printf("\n");
	  printf(" %f ", A.elements[i*A.width + j]);
	}
  printf("\n");
}

float cell(Matrix A, int row, int column) {
	return A.elements[row * A.width + column];
}

Matrix allocateMatrixToDevice(Matrix A) {
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc Matrix : %s\n", hipGetErrorString(err));
	err = hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy Matrix to device: %s\n",hipGetErrorString(err));
	return d_A;
}

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
  
  float Cvalue = 0.0;
  
  /* calculate value for C(row, column) */
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  
  /* not all threads in grid need return a value, as C may not fit exactly the grid */
  if (row > A.height || col > B.width) return;
  
  /* we are using Row Major representation for the matrix */
  for (int e = 0; e < A.width; ++e) {
	int a = row * A.width + e; /* row major, so just add e to index*/
	int b = e * B.width + col; /* row major, so multiply index by e */
    Cvalue += (A.elements[a] * B.elements[b]);
  }
  C.elements[row * C.width + col] = Cvalue;
}

void matmul(Matrix A, Matrix B, Matrix C) {

    /* copy the matrices to the GPU */
	Matrix d_A = allocateMatrixToDevice(A);
	Matrix d_B = allocateMatrixToDevice(B);
	Matrix d_C = allocateMatrixToDevice(C);
	
	/* specify 2 dimensional blocks of 16 x 16 = 256 threads per block */
	dim3 dimBlock(16,16);
	
	/* calculate how many blocks we need to perform the calculation */
	/* the grid is based on the size of the product matrix */
	/* ie: A(2,3) * B(3,4) = C(2,4) */
	/* A(height,width) * B(height,width) = C(A height, B width) */
	dim3 dimGrid(
	            ( (B.width + dimBlock.x - 1 ) / dimBlock.x),
				( (A.height + dimBlock.y -1 ) / dimBlock.y)
				);
	
	/* launch a grid and run the kernel function*/
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A,d_B,d_C);
	
	/* wait for all threads to finish */
	hipError_t err = hipDeviceSynchronize();
	
	err = hipMemcpy(C.elements, d_C.elements, C.height * C.width * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_A.elements);
	hipFree(d_B.elements);
}



int main(void)
{
  Matrix A = initMatrix(4,4);
  Matrix B = initMatrix(4,4);
  Matrix C = initMatrix(4,4);
  
  setRandom(A);
  setRandom(B);
  
  printMatrix(A);
  printMatrix(B);
  
  matmul(A,B,C);
  printMatrix(C);

  float c_0_0 = cell(A,0,0) * cell(B,0,0) + cell(A,0,1) * cell(B,1,0) + cell(A,0,2) * cell(B,2,0) + cell(A,0,3) * cell(B,3,0);
  printf("%f\n", c_0_0);
  assert(c_0_0 == cell(C,0,0));
  
}
